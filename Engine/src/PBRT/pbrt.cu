#include "hip/hip_runtime.h"
#include "pbrt.h"

#include "ray.h"
#include "BVH/boundingBox.h"
#include "intersection.h"
#include "Shape/sphere.h"
#include "Shape/square.h"
#include "Shape/cube.h"

#include "spectrum.h"
#include "Material/diffuseMaterial.h"

#include "Sampler/rng.h"
#include "Light/light.h"

#include <GL/glew.h>
#include <cuda_gl_interop.h>

namespace CudaPBRT
{
    CPU_GPU Shape* Create(const ShapeData& data)
    {
        switch (data.type)
        {
        case ShapeType::Sphere:
            return new Sphere(data);
        case ShapeType::Cube:
            return new Cube(data);
        case ShapeType::Square:
            return new Square(data);
        default:
            printf("Unknown ShapeType!\n");
            return nullptr;
        }
    }
    
    CPU_GPU Material* Create(const MaterialData& data)
    {
        switch (data.type)
        {
        case MaterialType::DiffuseReflection:
            return new DiffuseMaterial(data);
        default:
            printf("Unknown MaterialType!\n");
            return nullptr;
        }
    }

    CPU_GPU Light* Create(const LightData& data)
    {
        switch (data.type)
        {
        case LightType::ShapeLight:
            return new ShapeLight(data);
        default:
            printf("Unknown LightType!\n");
            return nullptr;
        }
    }

    CPU_GPU Ray CastRay(const PerspectiveCamera& camera, const glm::vec2& p)
    {
        glm::vec2 ndc = 2.f * p / glm::vec2(camera.width, camera.height);
        ndc.x = ndc.x - 1.f;
        ndc.y = 1.f - ndc.y;

        float aspect = static_cast<float>(camera.width) / static_cast<float>(camera.height);

        // point in camera space
        float radian = glm::radians(camera.fovy * 0.5f);
        glm::vec3 pCamera = glm::vec3(
            ndc.x * glm::tan(radian) * aspect,
            ndc.y * glm::tan(radian),
            1.f
        );

        Ray ray(glm::vec3(0), pCamera);

        ray.O = camera.position + ray.O.x * camera.right + ray.O.y * camera.up;
        ray.DIR = glm::normalize(ray.DIR.z * camera.forward +
                                 ray.DIR.y * camera.up +
                                 ray.DIR.x * camera.right);

        return ray;
    }

    CPU_GPU void writePixel(int iteration, float3& hdr_pixel, uchar4& pixel, const Spectrum& radiance)
    {
        glm::vec3 color(radiance);
        glm::vec3 preColor = glm::vec3(hdr_pixel.x, hdr_pixel.y, hdr_pixel.z);

        color = glm::mix(preColor, color, 1.f / float(iteration));
        
        hdr_pixel.x = color.x;
        hdr_pixel.y = color.y;
        hdr_pixel.z = color.z;

        // tone mapping
        color = color / (1.f + color);

        // gammar correction
        color = glm::pow(color, glm::vec3(1.f / 2.2f));

        color = glm::mix(glm::vec3(0.f), glm::vec3(255.f), color);

        color = glm::clamp(color, glm::vec3(0), glm::vec3(255));

        pixel.x = static_cast<int>(color.r);
        pixel.y = static_cast<int>(color.g);
        pixel.z = static_cast<int>(color.b);
        pixel.w = 255;
    }

    template<typename T, typename DataType>
    __global__ void CreateArray(T** device_array, DataType* data, size_t* max_count)
    {
        int id = blockIdx.x;
        if (id >= (*max_count))
        {
            return;
        }
        
        device_array[id] = Create(data[id]);
    }

    template<typename T>
    __global__ void FreeArray(T** device_array, size_t* max_count)
    {
        if (max_count == nullptr)
        {
            return;
        }

        for (int i = 0; i < *max_count; ++i)
        {
            if (device_array[i])
            {
                delete device_array[i];
                device_array[i] = nullptr;
            }
        }
    }
    

    template<typename T, typename DataType>
    void CreateArrayOnCude<T, DataType>(T**& dev_array, size_t*& dev_count, std::vector<DataType>& host_data)
    {
        DataType* device_data;
        size_t max_count = host_data.size();

        hipMalloc((void**)&dev_count, sizeof(size_t));
        CUDA_CHECK_ERROR();

        hipMemcpy(dev_count, &max_count, sizeof(size_t), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMalloc((void**)&device_data, sizeof(DataType) * max_count);
        CUDA_CHECK_ERROR();

        hipMemcpy(device_data, host_data.data(), sizeof(DataType) * max_count, hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMalloc((void**)&dev_array, sizeof(T*) * max_count);
        CUDA_CHECK_ERROR();

        // Launch a kernel on the GPU with one thread for each element.
        KernalConfig createConfig({ max_count, 1, 1 }, { 0, 0, 0 });
        CreateArray<T, DataType> << < createConfig.numBlocks, createConfig.threadPerBlock >> > (dev_array, device_data, dev_count);

        // hipDeviceSynchronize waits for the kernel to finish
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();

        CUDA_FREE(device_data);
    }

    template void CreateArrayOnCude<Light, LightData>(Light**& dev_array, size_t*& dev_count, std::vector<LightData>& data);
    template void CreateArrayOnCude<Shape, ShapeData>(Shape**& dev_array, size_t*& dev_count, std::vector<ShapeData>& data);
    template void CreateArrayOnCude<Material, MaterialData>(Material**& dev_array, size_t*& dev_count, std::vector<MaterialData>& data);

    template<typename T>
    void FreeArrayOnCuda(T**& device_array, size_t*& count)
    {
        if (count == nullptr || device_array == nullptr)
        {
            return;
        }

        KernalConfig freeConfig({ 1, 1, 1 }, { 0, 0, 0 });
        FreeArray<T> << <freeConfig.numBlocks, freeConfig.threadPerBlock >> > (device_array, count);
        CUDA_CHECK_ERROR();

        CUDA_FREE(device_array);
        CUDA_FREE(count);
        CUDA_CHECK_ERROR();
    }

    template void FreeArrayOnCuda(Shape**& device_array, size_t*& count);
    template void FreeArrayOnCuda(Material**& device_array, size_t*& count);
    template void FreeArrayOnCuda(Light**& device_array, size_t*& count);

    __global__ void Draw(int* iteration, PerspectiveCamera* camera, uchar4* img, float3* hdr_img, Shape** shapes, size_t* shape_count, Light** lights, size_t* light_count, Material** materials)
    {
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        int y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x >= camera->width || y >= camera->height) {
            return;
        }
        
        int index = x + (y * camera->width);
        
        CudaRNG rng(*iteration, index, 1);

        Ray ray = CastRay(*camera, {x + rng.rand(), y + rng.rand() });

        Spectrum radiance(0.f);
        Spectrum throughput(1.f);
        
        Intersection shape_intersection, light_intersection;

        int depth = 0;
        while (depth++ < MaxDepth)
        {
            // find itersection
            // TODO: use BVH for intersection testing
            shape_intersection.t = CudaPBRT::FloatMax;
            light_intersection.t = CudaPBRT::FloatMax;
            shape_intersection.id = -1;
            light_intersection.id = -1;

            for (int i = 0; i < (*light_count); ++i)
            {
                Intersection it;
                if (lights[i]->IntersectionP(ray, it) && it.t > 0.f && it.t < light_intersection.t)
                {
                    light_intersection = it;
                    light_intersection.id = i;
                }
            }
            for (int i = 0; i < (*shape_count); ++i)
            {
                Intersection it;
                if (shapes[i]->IntersectionP(ray, it) && it.t > 0.f && it.t < shape_intersection.t)
                {
                    shape_intersection = it;
                    shape_intersection.id = i;
                    shape_intersection.material_id = shapes[i]->material_id;
                }
            }

            if (light_intersection.id >= 0)
            {
                if (shape_intersection.id < 0 || (shape_intersection.id >= 0 && shape_intersection.t > light_intersection.t))
                {
                    // hit light source
                    throughput *= lights[light_intersection.id]->GetLe();
                    radiance += throughput;
                    break;
                }
            }
            else if (shape_intersection.id >= 0)
            {
                //radiance += materials[shape_intersection.material_id]->GetAlbedo();
                //radiance += 0.5f * (shape_intersection.normal + 1.f);
                //break;
                glm::vec3 point = ray * shape_intersection.t;
                BSDF& bsdf = materials[shape_intersection.material_id]->GetBSDF();

                glm::vec3 normal = glm::normalize(shape_intersection.normal);
                normal = materials[shape_intersection.material_id]->GetNormal(normal);

                BSDFSample bsdfSample = bsdf.Sample_f(materials[shape_intersection.material_id]->GetAlbedo(), -ray.DIR, normal, {rng.rand(), rng.rand()});

                if (bsdfSample.pdf == 0.f || glm::length(bsdfSample.f) == 0.f)
                {
                    break;
                }

                throughput *= bsdfSample.f * glm::abs(glm::dot(bsdfSample.wiW, normal)) / bsdfSample.pdf;
                
                ray = Ray::SpawnRay(point, bsdfSample.wiW);
            }
            else
            {
                break;
            }
        }

        writePixel(*iteration, hdr_img[y * camera->width + x], img[y * camera->width + x], radiance);
    }

    CudaPathTracer::CudaPathTracer()
    {

    }

    CudaPathTracer::~CudaPathTracer()
    {
        FreeCuda();
    }

    void CudaPathTracer::InitCuda(PerspectiveCamera& camera, int device)
    {
        // set basic properties
        width = camera.width;
        height = camera.height;
        
        m_Iteration = 1;

        // Choose which GPU to run on, change this on a multi-GPU system.
        hipSetDevice(device);;
        CUDA_CHECK_ERROR();

        glGenTextures(1, &m_DisplayImage);
        glBindTexture(GL_TEXTURE_2D, m_DisplayImage);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
        glBindTexture(GL_TEXTURE_2D, 0);

        // create rendered image on cpu
        host_image = new uchar4[width * height];

        // Create cuda device pointers
        // Allocate GPU buffers for three vectors (two input, one output).
        hipMalloc((void**)&device_iteration, sizeof(int));
        CUDA_CHECK_ERROR();     

        hipMalloc((void**)&device_camera, sizeof(PerspectiveCamera));
        CUDA_CHECK_ERROR();

        // Copy input vectors from host memory to GPU buffers.
        hipMemcpy(device_camera, &camera, sizeof(PerspectiveCamera), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMalloc((void**)&device_hdr_image, sizeof(float3) * width * height);
        CUDA_CHECK_ERROR();

        hipMalloc((void**)&device_image, sizeof(uchar4) * width * height);
        CUDA_CHECK_ERROR();
    }

    void CudaPathTracer::FreeCuda()
    {
        FreeArrayOnCuda<Shape>(device_shapes, device_shape_count);
        FreeArrayOnCuda<Material>(device_materials, device_material_count);
        FreeArrayOnCuda<Light>(device_lights, device_light_count);

        CUDA_FREE(device_camera);
        CUDA_FREE(device_image);
        CUDA_FREE(device_hdr_image);
        CUDA_FREE(device_iteration);

        if (host_image)
        {
            delete[] host_image;
        }
        if (m_DisplayImage)
        {
            glDeleteTextures(1, &m_DisplayImage);
        }
    }

    void CudaPathTracer::Run()
    {

        hipMemcpy(device_iteration, &m_Iteration, sizeof(int), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        KernalConfig drawConfig({width, height, 1}, {3, 3, 0});

        //glm::ivec2 blockSize(5, 5);
        //dim3 numBlocks(UpperBinary(width >> blockSize.x), UpperBinary(height >> blockSize.y), 1);
        //dim3 threadPerBlock(BIT(blockSize.x), BIT(blockSize.y), 1);

        // draw color to pixels
        Draw <<< drawConfig.numBlocks, drawConfig.threadPerBlock >>> (device_iteration, device_camera, device_image, device_hdr_image,
                                                                      device_shapes, device_shape_count, 
                                                                      device_lights, device_light_count,
                                                                      device_materials);

        ++m_Iteration;

        // wait GPU to finish computation
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR();

        // Copy rendered result to CPU.
        hipMemcpy(host_image, device_image, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);
        CUDA_CHECK_ERROR();

        // pass render result to glTexture2D
        glBindTexture(GL_TEXTURE_2D, m_DisplayImage);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void*)host_image);
        glBindTexture(GL_TEXTURE_2D, 0);
    }

    void CudaPathTracer::UpdateCamera(PerspectiveCamera& camera)
    {
        // Copy input vectors from host memory to GPU buffers.
        hipMemcpy(device_camera, &camera, sizeof(PerspectiveCamera), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();
    }
}