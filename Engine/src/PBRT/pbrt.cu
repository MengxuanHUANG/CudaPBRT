#include "hip/hip_runtime.h"
#include "pbrt.h"

#include "ray.h"
#include "BVH/boundingBox.h"
#include "intersection.h"
#include "Shape/sphere.h"
//#include "bsdf.h"
//#include "bxdfs.h"

#include <GL/glew.h>
#include <cuda_gl_interop.h>

#ifdef CUDA_PBRT_DEBUG
#define CUDA_CHECK_ERROR(state, message) if(state != hipSuccess) fprintf(stderr, message);
#else
#define CUDA_CHECK_ERROR(state, message) 
#endif

namespace CudaPBRT
{
    __global__ void CreateShapes(Shape** device_shapes, ShapeData* data, unsigned int* max_count)
    {
        int id = blockIdx.x;
        if (id >= *max_count)
        {
            return;
        }
        device_shapes[id] = new Sphere(data[id]);
    }
    
    __global__ void FreeShapes(Shape** device_shapes, unsigned int* max_count)
    {
        for (int i = 0; i < *max_count; ++i)
        {
            if (device_shapes[i])
            {
                delete device_shapes[i];
                device_shapes[i] = nullptr;
            }
        }
    }

    __global__ void NewPtr(TestCudaVirtual** tv)
    {
        (*tv) = new B();
        printf("new %X\n", (*tv));
        (*tv)->value.r = 255.f;
        (*tv)->value.g = -255.f;
    }

    __global__ void ReadPtr(TestCudaVirtual** tv)
    {
        //printf("read %X : (%f, %f, %f)\n",(*tv), (*tv)->value.x, (*tv)->value.y, (*tv)->value.z);
        printf("read %X : GetValue() {%f}\n", (*tv), (*tv)->GetValue());
    }

    __global__ void DeletePtr(TestCudaVirtual** tv)
    {
        printf("delete %X\n", (*tv));
        delete (*tv);
    }

    __global__ void Draw(PerspectiveCamera* camera, uchar4* img, Shape** shapes, unsigned int* shape_count)
    {
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        int y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x >= camera->width || y >= camera->height) {
            return;
        }
        //printf("Hello from block (%d, %d, %d), thread (%d, %d, %d), block dim (%d, %d, %d)\n",
        //    blockIdx.x, blockIdx.y, blockIdx.z,
        //    threadIdx.x, threadIdx.y, threadIdx.z,
        //    blockDim.x, blockDim.y, blockDim.z);

        glm::vec2 ndc = 2.f * (glm::vec2(x, y) / glm::vec2(camera->width, camera->height));
        ndc.x = ndc.x - 1.f;
        ndc.y = 1.f - ndc.y;

        float aspect = camera->width / camera->height;

        // point in camera space
        glm::vec3 pCamera = glm::vec3(
            ndc.x * glm::tan(camera->fovy * 0.5f) * aspect,
            ndc.y * glm::tan(camera->fovy * 0.5f),
            1.f
        );

        Ray ray(glm::vec3(0), pCamera);

        ray.O = camera->position + ray.O.x * camera->right + ray.O.y * camera->up;
        ray.DIR = glm::normalize(ray.DIR.z * camera->forward +
                                 ray.DIR.y * camera->up +
                                 ray.DIR.x * camera->right);

        glm::vec3 color;

        // display normal
        //color = 0.5f * (ray.DIR + 1.f);

        color = glm::vec3(0.f);
        Intersection intersection;
        intersection.t = 10000.f;

        for (int i = 0; i < *shape_count; ++i)
        {
            Intersection it;

            if (shapes[i]->IntersectionP(ray, it) && it.t < intersection.t)
            {
                intersection = it;
                intersection.id = i;
            }
        }
        
        if (intersection.id >= 0)
        {
            color = 0.5f * (intersection.normal + 1.f);
        }
        
        // tone mapping
        //color = color / (1.f + color);

        // gammar correction
        //color = glm::pow(color, glm::vec3(1.f / 2.2f));

        img[y * camera->width + x].x = static_cast<int>(glm::mix(0.f, 255.f, color.x));
        img[y * camera->width + x].y = static_cast<int>(glm::mix(0.f, 255.f, color.y));
        img[y * camera->width + x].z = static_cast<int>(glm::mix(0.f, 255.f, color.z));
        img[y * camera->width + x].w = 255;
    }

    CudaPathTracer::CudaPathTracer()
    {

    }

    CudaPathTracer::~CudaPathTracer()
    {

    }

    void CudaPathTracer::InitCuda(PerspectiveCamera& camera, int device)
    {
        // set basic properties
        width = camera.width;
        height = camera.height;

        // Choose which GPU to run on, change this on a multi-GPU system.
        hipError_t cudaStatus = hipSetDevice(device);;
        CUDA_CHECK_ERROR(cudaStatus, "hipSetDevice failed!");

        glGenTextures(1, &m_DisplayImage);
        glBindTexture(GL_TEXTURE_2D, m_DisplayImage);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
        glBindTexture(GL_TEXTURE_2D, 0);

        // create rendered image on cpu
        host_image = new uchar4[width * height];

        // Create cuda device pointers
        // Allocate GPU buffers for three vectors (two input, one output).
        cudaStatus = hipMalloc((void**)&device_camera, sizeof(PerspectiveCamera));
        CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!");

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(device_camera, &camera, sizeof(PerspectiveCamera), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy host to device failed!");

        cudaStatus = hipMalloc((void**)&device_image, sizeof(uchar4) * width * height);
        CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!");
    }

    void CudaPathTracer::CreateShapesOnCuda(std::vector<ShapeData>& shapeData)
    {
        hipError_t cudaStatus;

        ShapeData* device_shapeData;
        unsigned int max_count = shapeData.size();

        cudaStatus = hipMalloc((void**)&device_shape_count, sizeof(unsigned int));
        CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!");

        cudaStatus = hipMemcpy(device_shape_count, &max_count, sizeof(unsigned int), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy host to device failed!");

        cudaStatus = hipMalloc((void**)&device_shapeData, sizeof(ShapeData) * max_count);
        CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!");

        cudaStatus = hipMemcpy(device_shapeData, shapeData.data(), sizeof(ShapeData) * max_count, hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy host to device failed!");

        cudaStatus = hipMalloc((void**)&device_shapes, sizeof(Shape*) * max_count);
        CUDA_CHECK_ERROR(cudaStatus, "hipMalloc failed!");

        // Launch a kernel on the GPU with one thread for each element.
        CreateShapes <<< max_count, 1 >>> (device_shapes, device_shapeData, device_shape_count);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        CUDA_CHECK_ERROR(cudaStatus, "cuda launch failed!");

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
        }

        if (device_shapeData != nullptr)
        {
            hipFree(device_shapeData);
        }
    }

    void CudaPathTracer::FreeShapesOnCuda()
    {
        hipError_t cudaStatus;

        // Launch a kernel on the GPU with one thread for each element.
        FreeShapes <<<1, 1>>> (device_shapes, device_shape_count);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        CUDA_CHECK_ERROR(cudaStatus, "cuda launch failed!");

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
        }
    }

    void CudaPathTracer::FreeCuda()
    {
        if (device_camera != nullptr)
        {
            hipFree(device_camera);
            device_camera = nullptr;
        }
        if (device_image != nullptr)
        {
            hipFree(device_image);
            device_image = nullptr;
        }
        if (device_shapes != nullptr)
        {
            hipFree(device_shapes);
            device_shapes = nullptr;
        }
        if (device_shape_count != nullptr)
        {
            hipFree(device_shape_count);
        }

        if (host_image)
        {
            delete[] host_image;
        }
        if (m_DisplayImage)
        {
            glDeleteTextures(1, &m_DisplayImage);
        }
    }

    void CudaPathTracer::Run()
    {
        hipError_t cudaStatus;

        dim3 numBlocks(UpperBinary(width >> 4), UpperBinary(height >> 4), 1);
        dim3 threadPerBlock(16, 16, 1);

        // Launch a kernel on the GPU with one thread for each element.
        Draw <<< numBlocks, threadPerBlock >>> (device_camera, device_image, device_shapes, device_shape_count);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        CUDA_CHECK_ERROR(cudaStatus, "cuda launch failed!");

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
        }

        // Copy rendered result to CPU.
        cudaStatus = hipMemcpy(host_image, device_image, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);
        CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy from device to host failed!");

        // pass render result to glTexture2D
        glBindTexture(GL_TEXTURE_2D, m_DisplayImage);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void*)host_image);
        glBindTexture(GL_TEXTURE_2D, 0);
    }

    void CudaPathTracer::UpdateCamera(PerspectiveCamera& camera)
    {
        // Copy input vectors from host memory to GPU buffers.
        hipError_t cudaStatus = hipMemcpy(device_camera, &camera, sizeof(PerspectiveCamera), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR(cudaStatus, "hipMemcpy host to device failed!");
    }
}